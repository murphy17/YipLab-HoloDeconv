#include "hip/hip_runtime.h"

/*
 *
 * Proof-of-concept for GPU holographic deconvolution.
 * Michael Murphy, May 2017
 * Yip Lab
 *
 */

// this is the fastest yet on Tegra, keeping PSF on device gives fastest yet on Titan
// ... consumes a TON of memory. immediately fills the Tegra
// could serialize batches of slices, I suppose - hiding latency of multiply is main thing
// ... I've got an experiment for the multiply in the works, would require transposing the PSF cube though

#include <opencv2/opencv.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/gpu/gpu.hpp>
//#include <opencv2/core/cuda.hpp>
#include <hip/hip_runtime.h>
#include <hipfft/hipfftXt.h>
#include <algorithm>

#include "common.h"

#define N 1024
#define LOG2N 10
#define DX (5.32f / 1024.f)
#define DY (5.32f / 1024.f) // (6.66f / 1280.f) ... are these supposed to be the *SAME*? very close, but exact same helps a lot!
#define LAMBDA0 0.000488f
#define NUM_SLICES 100

typedef unsigned char byte;

// Convenience method for plotting
void imshow(cv::Mat in)
{
	cv::namedWindow("Display window", cv::WINDOW_NORMAL); // Create a window for display.
	cv::Mat out = in;
	hipDeviceSynchronize();
	if (out.channels() == 2)
	{
		cv::Mat channels[2];
		cv::split(out, channels);
		cv::magnitude(channels[0], channels[1], out);
	}
	out.convertTo(out, CV_32FC1);
	cv::normalize(out, out, 1.0, 0.0, cv::NORM_MINMAX, -1);
	cv::imshow("Display window", out); // Show our image inside it.
	cv::waitKey(0);
}

void imshow(cv::gpu::GpuMat in) //, bool log=false)
{
	cv::namedWindow("Display window", cv::WINDOW_NORMAL); // Create a window for display.
	cv::Mat out;
	hipDeviceSynchronize();
	in.download(out);
	if (out.channels() == 2)
	{
		cv::Mat channels[2];
		cv::split(out, channels);
		cv::magnitude(channels[0], channels[1], out);
	}
	out.convertTo(out, CV_32FC1);
//	if (log)
//		cv::log(out, out);
	cv::normalize(out, out, 1.0, 0.0, cv::NORM_MINMAX, -1);
	cv::imshow("Display window", out); // Show our image inside it.
	cv::waitKey(0);
}

// Kernel to construct the point-spread function at distance z.
// exploits 4-fold symmetry (PSF is also radially symmetric, but that's harder...)
// note that answer is scaled between +/-1
__global__
void construct_psf(float z, hipfftComplex *g, float norm)
{
	const int i = blockIdx.x;
	const int j = threadIdx.x; // blockDim shall equal N

//	const int ii = (N - 1) - i;
//	const int jj = (N - 1) - j;

	// 'FFT-even symmetry' - periodic extension must be symmetric about (0,0)
	float x = (i - N/2) * DX;
	float y = (j - N/2) * DY;

	// could omit negation here, symmetries of trig functions take care of it
	float r = (-2.f / LAMBDA0) * norm3df(x, y, z);

	// exp(ix) = cos(x) + isin(x)
	float re, im;
	sincospif(r, &im, &re);

	// numerical conditioning, important for half-precision FFT
	// also corrects the sign flip above
	r = __fdividef(r, norm); // norm = -2.f * z / LAMBDA0

	// re(iz) = -im(z), im(iz) = re(z)
	hipfftComplex g_ij;
	g_ij.x = __fdividef(-im, r); // im, r);
	g_ij.y = __fdividef(re, r);

	// CUDA takes care of coalescing the reversed access, this is fine
	g[i*N+j] = g_ij;
//	g[i*N+jj] = g_ij;
//	g[ii*N+j] = g_ij;
//	g[ii*N+jj] = g_ij;
}

// exploit Fourier duality to shift without copying
// credit to http://www.orangeowlsolutions.com/archives/251
__global__
void frequency_shift(hipfftComplex *data)
{
    const int i = blockIdx.x;
    const int j = threadIdx.x;

	const float a = 1 - 2 * ((i+j) & 1); // this looks like a checkerboard?

	data[i*N+j].x *= a;
	data[i*N+j].y *= a;
}

__device__ __forceinline__
hipfftComplex _mul(hipfftComplex a, hipfftComplex b)
{
	hipfftComplex c;

	// with O3 on this is fine
	c.x = a.x * b.x - a.y * b.y;
	c.y = a.x * b.y + a.y * b.x;

	return c;
}

__global__
void batch_multiply(hipfftComplex *z, const __restrict__ hipfftComplex *w)
{
	const int i = blockIdx.x;
	const int j = threadIdx.x;

	hipfftComplex w_ij = w[i*N+j];

	for (int k = 0; k < NUM_SLICES; k++)
	{
		z[i*N+j] = _mul(z[i*N+j], w_ij);

		z += N*N;
	}
}

__global__
//__device__
void quadrant_multiply(hipfftComplex *z, const __restrict__ hipfftComplex *w) //, int i, int j)
{
	const int i = blockIdx.x;
	const int j = threadIdx.x;
	const int ii = N-i;
	const int jj = N-j;

	// this saves 8 registers (does it still?)
	int cond = 0;
	if (i>0&&i<N/2) cond |= 1;
	if (j>0&&j<N/2) cond |= 2;

	hipfftComplex w_[4];
	w_[0] = w[i*N+j];
	if (cond & 1) w_[1] = w[ii*N+j];
	if (cond & 2) w_[2] = w[i*N+jj];
	if (cond == 3) w_[3] = w[ii*N+jj];

	hipfftComplex z_ij;

	// conditional unwrapping
	// this had no effect, but compiler didn't seem to be doing it?
	switch (cond)
	{
		case 3:
		for (int k = 0; k < NUM_SLICES; k++)
		{
			z_ij = z[i*N+j];
			z[i*N+j] = _mul(w_[0], z_ij);
			z[ii*N+j] = _mul(w_[1], z_ij);
			z[i*N+jj] = _mul(w_[2], z_ij);
			z[ii*N+jj] = _mul(w_[3], z_ij);
			z += N*N;
		}
		break;

		case 2:
		for (int k = 0; k < NUM_SLICES; k++)
		{
			z_ij = z[i*N+j];
			z[i*N+j] = _mul(w_[0], z_ij);
			z[i*N+jj] = _mul(w_[2], z_ij);
			z += N*N;
		}
		break;

		case 1:
		for (int k = 0; k < NUM_SLICES; k++)
		{
			z_ij = z[i*N+j];
			z[i*N+j] = _mul(w_[0], z_ij);
			z[ii*N+j] = _mul(w_[1], z_ij);
			z += N*N;
		}
		break;

		case 0:
		for (int k = 0; k < NUM_SLICES; k++)
		{
			z_ij = z[i*N+j];
			z[i*N+j] = _mul(w_[0], z_ij);
			z += N*N;
		}
		break;
	}
}

//__global__
//void quadrant_multiply(hipfftComplex *z, const __restrict__ hipfftComplex *w)
//{
//	const int i = blockIdx.x;
//	const int j = threadIdx.x;
//
//	// permits using nicely-sized kernel dimensions
//	_quadrant_multiply(z, w, i, j);
//	if (i == N/2-1) _quadrant_multiply(z, w, i+1, j);
//	if (j == N/2-1) _quadrant_multiply(z, w, i, j+1);s
//	if (i == N/2-1 && j == N/2-1) _quadrant_multiply(z, w, i+1, j+1);
//}

__global__
void mirror_quadrants(hipfftComplex *z)
{
	const int i = blockIdx.x;
	const int j = threadIdx.x;
	const int ii = N-i;
	const int jj = N-j;

	if (j>0&&j<N/2) z[i*N+jj] = z[i*N+j];
	if (i>0&&i<N/2) z[ii*N+j] = z[i*N+j];
	if (i>0&&i<N/2&&j>0&&j<N/2) z[ii*N+jj] = z[i*N+j];
}

__global__
void byte_to_complex(byte *b, hipfftComplex *z)
{
	const int i = blockIdx.x;
	const int j = threadIdx.x; // blockDim shall equal N

	z[i*N+j].x = ((float)(b[i*N+j])) / 255.f;
	z[i*N+j].y = 0.f;
}

__device__ __forceinline__
float _mod(hipfftComplex z)
{
	return hypotf(z.x, z.y);
}

__global__
void complex_modulus(hipfftComplex *z, float *r)
{
	const int i = blockIdx.x;
	const int j = threadIdx.x; // blockDim shall equal N

	for (int slice = 0; slice < NUM_SLICES; slice++)
	{
		r[i*N+j] = hypotf(z[i*N+j].x, z[i*N+j].y);

		z += N*N;
		r += N*N;
	}
}

__global__
void copy_buffer(hipfftComplex *a, hipfftComplex *b)
{
	const int i = blockIdx.x;
	const int j = threadIdx.x; // blockDim shall equal N

	b[i*N+j] = a[i*N+j];
}

hipError_t transfer_psf(hipfftComplex *psf, hipfftComplex *buffer, hipStream_t stream)
{
	hipMemcpy3DParms p = { 0 };
	p.srcPtr.ptr = psf;
	p.srcPtr.pitch = (N/2+1) * sizeof(hipfftComplex);
	p.srcPtr.xsize = (N/2+1);
	p.srcPtr.ysize = (N/2+1);
	p.dstPtr.ptr = buffer;
	p.dstPtr.pitch = N * sizeof(hipfftComplex);
	p.dstPtr.xsize = N;
	p.dstPtr.ysize = N;
	p.extent.width = (N/2+1) * sizeof(hipfftComplex);
	p.extent.height = (N/2+1);
	p.extent.depth = NUM_SLICES;
	p.kind = hipMemcpyHostToDevice;

	return hipMemcpy3DAsync(&p, stream);
}

int main(int argc, char* argv[])
{
	checkCudaErrors( hipDeviceReset() );

	int num_frames = 5;
	float z_min = 30;
	float z_step = 1;

	long long dims[] = {N, N};
	size_t work_sizes = 0;

	hipfftComplex *image;
	checkCudaErrors( hipMalloc((void **)&image, N*N*sizeof(hipfftComplex)) );
	hipfftComplex *psf;
	checkCudaErrors( hipMalloc((void **)&psf, N*N*sizeof(hipfftComplex)) );

	// allocate this on the host - that way CPU can manage transfer, not GPU, lets it run in async
	// (this is 3x slower on Titan, but faster on Tegra - recall that host and GPU memory are the same thing in Tegra)
	hipfftComplex *host_psf;
	checkCudaErrors( hipHostMalloc((void **)&host_psf, NUM_SLICES*(N/2+1)*(N/2+1)*sizeof(hipfftComplex)) );

	byte *image_u8;
	checkCudaErrors( hipMalloc((void **)&image_u8, N*N*sizeof(byte)) );

	hipStream_t math_stream, copy_stream;
	checkCudaErrors( hipStreamCreate(&math_stream) );
	checkCudaErrors( hipStreamCreate(&copy_stream) );

	hipfftComplex *in_buffers[2];
	checkCudaErrors( hipMalloc((void **)&in_buffers[0], NUM_SLICES*N*N*sizeof(hipfftComplex)) );
	checkCudaErrors( hipMalloc((void **)&in_buffers[1], NUM_SLICES*N*N*sizeof(hipfftComplex)) );

	float *out_buffer;
	checkCudaErrors( hipMalloc((void **)&out_buffer, NUM_SLICES*N*N*sizeof(float)) );

	hipfftHandle fft_plan;
	checkCudaErrors( hipfftCreate(&fft_plan) );
	checkCudaErrors( hipfftXtMakePlanMany( \
			fft_plan, 2, dims, \
			NULL, 1, 0, HIP_C_32F, \
			NULL, 1, 0, HIP_C_32F, \
			1, &work_sizes, HIP_C_32F) );
	checkCudaErrors( hipfftSetStream(fft_plan, math_stream) );

	// cache 1/4 of the PSF (could do 1/8th too)
	for (int slice = 0; slice < NUM_SLICES; slice++)
	{
		float z = z_min + z_step * slice;

//		checkCudaErrors( hipMemset(psf, 0, N*N*sizeof(hipfftComplex)) ); // make sure works fine without this

		// generate the PSF, weakly taking advantage of symmetry to speed up
		construct_psf<<<N/2+1, N/2+1>>>(z, psf, -2.f * z / LAMBDA0);

		// testing symmetry
		mirror_quadrants<<<N/2+1, N/2+1>>>(psf);

		// FFT in-place
		checkCudaErrors( hipfftXtExec(fft_plan, psf, psf, HIPFFT_FORWARD) );
		checkCudaErrors( hipStreamSynchronize(math_stream) );

		// testing symmetry
		// mirror_quadrants<<<N/2+1, N/2+1, 0, streams[0]>>>(psf);

		// do the frequency shift here instead, complex multiplication commutes
		// this is subtle - shifting in conjugate domain means we don't need to FFT shift later
		frequency_shift<<<N/2+1, N/2+1>>>(psf);

		// TODO: the PSF quadrants themselves are symmetric matrices...

		// copy the upper-left submatrix
		checkCudaErrors( hipMemcpy2D( \
				host_psf + (N/2+1)*(N/2+1)*slice, (N/2+1)*sizeof(hipfftComplex), \
				psf, N*sizeof(hipfftComplex), \
				(N/2+1)*sizeof(hipfftComplex), N/2+1, \
				hipMemcpyDeviceToHost \
				) );
	}

	// preemptively load PSF for the first frame
	checkCudaErrors( transfer_psf(host_psf, in_buffers[0], copy_stream) );
	checkCudaErrors( hipStreamSynchronize(copy_stream) );

	volatile bool frameReady = true; // this would be updated by the camera

	// this would be a copy from a frame buffer on the Tegra
	cv::Mat A = cv::imread("test_square.bmp", CV_LOAD_IMAGE_GRAYSCALE);

	for (int frame = 0; frame < num_frames; frame++)
	{
		hipfftComplex *in_buffer = in_buffers[frame % 2];

		// wait for a frame...
		while (!frameReady) { ; }
		// ... and copy
		checkCudaErrors( hipMemcpyAsync(image_u8, A.data, N*N*sizeof(byte), hipMemcpyHostToDevice, math_stream) );

		// queue transfer for next frame, waiting for it to finish if necessary(?)
		checkCudaErrors( hipStreamSynchronize(math_stream) );
		checkCudaErrors( hipStreamSynchronize(copy_stream) );
		checkCudaErrors( transfer_psf(host_psf, in_buffers[(frame + 1) % 2], copy_stream) );

		// up-cast to complex
		byte_to_complex<<<N, N, 0, math_stream>>>(image_u8, image);

		// FFT the image in-place
		checkCudaErrors( hipfftXtExec(fft_plan, image, image, HIPFFT_FORWARD) );

		// random thought: an abstraction layer between kernel allocation and matrix dims would be nice
		// will likely involve template method

//		for (int slice = 0; slice < NUM_SLICES; slice++)
//		{
//			imshow(cv::gpu::GpuMat(N, N, CV_32FC2, in_buffer + N*N*slice), false);
//		}

		// batch-multiply with FFT'ed image
		// TODO: write a wrapper that takes care of ugly dimension sizes
		quadrant_multiply<<<N/2+1, N/2+1, 0, math_stream>>>(in_buffer, image);

		// inverse FFT that product
		// I have yet to see any speedup from batching the FFTs
		for (int slice = 0; slice < NUM_SLICES; slice++)
		{
			checkCudaErrors( hipfftXtExec(fft_plan, in_buffer + N*N*slice, in_buffer + N*N*slice, HIPFFT_BACKWARD) );
		}

		// complex modulus - faster to loop outside kernel, for some reason
		complex_modulus<<<N, N, 0, math_stream>>>(in_buffer, out_buffer);

		// start timer after first run, GPU "warmup"
		if (frame == 0)
			cudaTimerStart();
	}

	checkCudaErrors( hipDeviceSynchronize() );

	std::cout << cudaTimerStop() / (num_frames - 1) << "ms" << std::endl;

	if (argc == 2)
	{
		for (int slice = 0; slice < NUM_SLICES; slice++)
		{
			imshow(cv::gpu::GpuMat(N, N, CV_32FC1, out_buffer + N*N*slice));
		}
	}

	// TODO: reimplement cleanup code once satisfied with implementation

	return 0;
}
