#include "hip/hip_runtime.h"

/*
 *
 * Proof-of-concept for GPU holographic deconvolution.
 * Michael Murphy, May 2017
 * Yip Lab
 *
 */

#include <opencv2/opencv.hpp>
#include <opencv2/highgui.hpp>
#include <hip/hip_runtime.h>
#include <hipfft/hipfftXt.h>
#include <algorithm>

#include "common.h"
#include "cuda_half.hpp"
#include "util.hpp"

#define FP32
//#define FP16

#define N 1024
#define DX (5.32f / 1024.f)
#define DY (6.66f / 1280.f)
#define LAMBDA0 0.000488f
#define NUM_SLICES 100

#ifdef FP32
typedef hipfftComplex complex;
typedef float real;
#endif
#ifdef FP16
typedef half2 complex;
typedef half real;
#endif

typedef unsigned char byte;

// Kernel to construct the point-spread function at distance z.
// exploits 4-fold symmetry (PSF is also radially symmetric, but that's harder...)
// note that answer is scaled between +/-1
__global__
void construct_psf(float z, complex *g, float norm)
{
	const int i = blockIdx.x;
	const int j = threadIdx.x; // blockDim shall equal N

	// 'FFT-even symmetry' - periodic extension must be symmetric about (0,0)
	float x = (i - N/2) * DX;
	float y = (j - N/2) * DY;

	// could omit negation here, symmetries of trig functions take care of it
	float r = (-2.f / LAMBDA0) * norm3df(x, y, z);

	// exp(ix) = cos(x) + isin(x)
	float re, im;
	sincospif(r, &im, &re);

	// numerical conditioning, important for half-precision FFT
	// also corrects the sign flip above
	r = __fdividef(r, norm); // norm = -2.f * z / LAMBDA0

	// re(iz) = -im(z), im(iz) = re(z)
	complex g_ij;
	g_ij.x = __fdividef(-im, r); // im, r);
	g_ij.y = __fdividef(re, r);

	// CUDA takes care of coalescing the reversed access, this is fine
	g[i*N+j] = g_ij;
}

// exploit Fourier duality to shift without copying
// credit to http://www.orangeowlsolutions.com/archives/251
template <class T>
__global__
void frequency_shift(T *data)
{
    const int i = blockIdx.x;
    const int j = threadIdx.x;

	const float a = 1 - 2 * ((i+j) & 1); // this looks like a checkerboard?

	data[i*N+j].x *= a;
	data[i*N+j].y *= a;
}

template <class T>
__device__ __forceinline__
T _mul(T a, T b)
{
	T c;

	c.x = a.x * b.x - a.y * b.y;
	c.y = a.x * b.y + a.y * b.x;

	return c;
}

__global__
void batch_multiply(complex *z, const __restrict__ complex *w)
{
	const int i = blockIdx.x;
	const int j = threadIdx.x;

	complex w_ij = w[i*N+j];

	for (int k = 0; k < NUM_SLICES; k++)
	{
		z[i*N+j] = _mul(z[i*N+j], w_ij);

		z += N*N;
	}
}

__global__
//__device__
void quadrant_multiply(complex *z, const __restrict__ complex *w) //, int i, int j)
{
	const int i = blockIdx.x;
	const int j = threadIdx.x;
	const int ii = N-i;
	const int jj = N-j;

	// this saves 8 registers (does it still?)
	int cond = 0;
	if (i>0&&i<N/2) cond |= 1;
	if (j>0&&j<N/2) cond |= 2;

	complex w_[4];
	w_[0] = w[i*N+j];
	if (cond & 1) w_[1] = w[ii*N+j];
	if (cond & 2) w_[2] = w[i*N+jj];
	if (cond == 3) w_[3] = w[ii*N+jj];

	complex z_ij;

	// conditional unwrapping
	// this had no effect, but compiler didn't seem to be doing it?
	switch (cond)
	{
		case 3:
		for (int k = 0; k < NUM_SLICES; k++)
		{
			z_ij = z[i*N+j];
			z[i*N+j] = _mul(w_[0], z_ij);
			z[ii*N+j] = _mul(w_[1], z_ij);
			z[i*N+jj] = _mul(w_[2], z_ij);
			z[ii*N+jj] = _mul(w_[3], z_ij);
			z += N*N;
		}
		break;

		case 2:
		for (int k = 0; k < NUM_SLICES; k++)
		{
			z_ij = z[i*N+j];
			z[i*N+j] = _mul(w_[0], z_ij);
			z[i*N+jj] = _mul(w_[2], z_ij);
			z += N*N;
		}
		break;

		case 1:
		for (int k = 0; k < NUM_SLICES; k++)
		{
			z_ij = z[i*N+j];
			z[i*N+j] = _mul(w_[0], z_ij);
			z[ii*N+j] = _mul(w_[1], z_ij);
			z += N*N;
		}
		break;

		case 0:
		for (int k = 0; k < NUM_SLICES; k++)
		{
			z_ij = z[i*N+j];
			z[i*N+j] = _mul(w_[0], z_ij);
			z += N*N;
		}
		break;
	}
}

//__global__
//void quadrant_multiply(complex *z, const __restrict__ complex *w)
//{
//	const int i = blockIdx.x;
//	const int j = threadIdx.x;
//
//	// permits using nicely-sized kernel dimensions
//	_quadrant_multiply(z, w, i, j);
//	if (i == N/2-1) _quadrant_multiply(z, w, i+1, j);
//	if (j == N/2-1) _quadrant_multiply(z, w, i, j+1);s
//	if (i == N/2-1 && j == N/2-1) _quadrant_multiply(z, w, i+1, j+1);
//}

__global__
void mirror_quadrants(complex *z)
{
	const int i = blockIdx.x;
	const int j = threadIdx.x;
	const int ii = N-i;
	const int jj = N-j;

	if (j>0&&j<N/2) z[i*N+jj] = z[i*N+j];
	if (i>0&&i<N/2) z[ii*N+j] = z[i*N+j];
	if (i>0&&i<N/2&&j>0&&j<N/2) z[ii*N+jj] = z[i*N+j];
}

__global__
void byte_to_complex(byte *b, complex *z)
{
	const int i = blockIdx.x;
	const int j = threadIdx.x; // blockDim shall equal N

	z[i*N+j].x = ((float)(b[i*N+j])) / 255.f;
	z[i*N+j].y = 0.f;
}

__device__ __forceinline__
float _mod(complex z)
{
	return hypotf(z.x, z.y);
}

__global__
void complex_modulus(complex *z, float *r)
{
	const int i = blockIdx.x;
	const int j = threadIdx.x; // blockDim shall equal N

	for (int slice = 0; slice < NUM_SLICES; slice++)
	{
		r[i*N+j] = hypotf(z[i*N+j].x, z[i*N+j].y);

		z += N*N;
		r += N*N;
	}
}

__global__
void copy_buffer(complex *a, complex *b)
{
	const int i = blockIdx.x;
	const int j = threadIdx.x; // blockDim shall equal N

	b[i*N+j] = a[i*N+j];
}

hipError_t transfer_psf(complex *psf, complex *buffer, hipStream_t stream)
{
	hipMemcpy3DParms p = { 0 };
	p.srcPtr.ptr = psf;
	p.srcPtr.pitch = (N/2+1) * sizeof(complex);
	p.srcPtr.xsize = (N/2+1);
	p.srcPtr.ysize = (N/2+1);
	p.dstPtr.ptr = buffer;
	p.dstPtr.pitch = N * sizeof(complex);
	p.dstPtr.xsize = N;
	p.dstPtr.ysize = N;
	p.extent.width = (N/2+1) * sizeof(complex);
	p.extent.height = (N/2+1);
	p.extent.depth = NUM_SLICES;
	p.kind = hipMemcpyHostToDevice;

	return hipMemcpy3DAsync(&p, stream);
}

int main(int argc, char* argv[])
{
	checkCudaErrors( hipDeviceReset() );


	int num_frames = 3;
	float z_min = 30;
	float z_step = 1;

	long long dims[] = {N, N};
	size_t work_sizes = 0;

	complex *image;
	checkCudaErrors( hipMalloc((void **)&image, N*N*sizeof(complex)) );
	complex *psf;
	checkCudaErrors( hipMalloc((void **)&psf, N*N*sizeof(complex)) );

	complex *host_psf;
	checkCudaErrors( hipHostMalloc((void **)&host_psf, NUM_SLICES*(N/2+1)*(N/2+1)*sizeof(complex)) );

	byte *image_u8;
	checkCudaErrors( hipMalloc((void **)&image_u8, N*N*sizeof(byte)) );

	hipStream_t math_stream, copy_stream;
	checkCudaErrors( hipStreamCreate(&math_stream) );
	checkCudaErrors( hipStreamCreate(&copy_stream) );

	complex *in_buffers[2];
	checkCudaErrors( hipMalloc((void **)&in_buffers[0], NUM_SLICES*N*N*sizeof(complex)) );
	checkCudaErrors( hipMalloc((void **)&in_buffers[1], NUM_SLICES*N*N*sizeof(complex)) );

	float *out_buffer;
	checkCudaErrors( hipMalloc((void **)&out_buffer, NUM_SLICES*N*N*sizeof(float)) );

	hipfftHandle fft_plan;
	checkCudaErrors( hipfftCreate(&fft_plan) );
	checkCudaErrors( hipfftXtMakePlanMany( \
			fft_plan, 2, dims, \
			NULL, 1, 0, HIP_C_32F, \
			NULL, 1, 0, HIP_C_32F, \
			1, &work_sizes, HIP_C_32F) );
	checkCudaErrors( hipfftSetStream(fft_plan, math_stream) );

	// cache 1/4 of the PSF (could do 1/8th too)
	for (int slice = 0; slice < NUM_SLICES; slice++)
	{
		float z = z_min + z_step * slice;

//		checkCudaErrors( hipMemset(psf, 0, N*N*sizeof(complex)) ); // make sure works fine without this

		// generate the PSF, weakly taking advantage of symmetry to speed up
		construct_psf<<<N/2+1, N/2+1>>>(z, psf, -2.f * z / LAMBDA0);
		mirror_quadrants<<<N/2+1, N/2+1>>>(psf);

		// FFT in-place
		checkCudaErrors( hipfftXtExec(fft_plan, psf, psf, HIPFFT_FORWARD) );
		checkCudaErrors( hipStreamSynchronize(math_stream) );

		// do the frequency shift here instead, complex multiplication commutes
		// this is subtle - shifting in conjugate domain means we don't need to FFT shift later
		frequency_shift<<<N/2+1, N/2+1>>>(psf);

		// TODO: the PSF quadrants themselves are symmetric matrices...

		// copy the upper-left submatrix
		checkCudaErrors( hipMemcpy2D( \
				host_psf + (N/2+1)*(N/2+1)*slice, (N/2+1)*sizeof(complex), \
				psf, N*sizeof(complex), \
				(N/2+1)*sizeof(complex), N/2+1, \
				hipMemcpyDeviceToHost \
				) );
	}

	// preemptively load PSF for the first frame
	checkCudaErrors( transfer_psf(host_psf, in_buffers[0], copy_stream) );
	checkCudaErrors( hipStreamSynchronize(copy_stream) );

	volatile bool frameReady = true; // this would be updated by the camera

	// this would be a copy from a frame buffer on the Tegra
	cv::Mat A = cv::imread("test_square.bmp", CV_LOAD_IMAGE_GRAYSCALE);

	for (int frame = 0; frame < num_frames; frame++)
	{
		complex *in_buffer = in_buffers[frame % 2];

		// wait for a frame...
		while (!frameReady) { ; }
		// ... and copy
		checkCudaErrors( hipMemcpyAsync(image_u8, A.data, N*N*sizeof(byte), hipMemcpyHostToDevice, math_stream) );

		// queue transfer for next frame, waiting for it to finish if necessary(?)
		checkCudaErrors( hipStreamSynchronize(math_stream) );
		checkCudaErrors( hipStreamSynchronize(copy_stream) );
		checkCudaErrors( transfer_psf(host_psf, in_buffers[(frame + 1) % 2], copy_stream) );

		// up-cast to complex
		byte_to_complex<<<N, N, 0, math_stream>>>(image_u8, image);

		// FFT the image in-place
		checkCudaErrors( hipfftXtExec(fft_plan, image, image, HIPFFT_FORWARD) );

		// random thought: an abstraction layer between kernel allocation and matrix dims would be nice
		// will likely involve template method

//		for (int slice = 0; slice < NUM_SLICES; slice++)
//		{
//			imshow(cv_gpu::GpuMat(N, N, CV_32FC2, in_buffer + N*N*slice), false);
//		}

		// batch-multiply with FFT'ed image
		// TODO: write a wrapper that takes care of ugly dimension sizes
		quadrant_multiply<<<N/2+1, N/2+1, 0, math_stream>>>(in_buffer, image);

		// inverse FFT that product
		// I have yet to see any speedup from batching the FFTs
		for (int slice = 0; slice < NUM_SLICES; slice++)
		{
			checkCudaErrors( hipfftXtExec(fft_plan, in_buffer + N*N*slice, in_buffer + N*N*slice, HIPFFT_BACKWARD) );
		}

		// complex modulus - faster to loop outside kernel, for some reason
		complex_modulus<<<N, N, 0, math_stream>>>(in_buffer, out_buffer);

		// start timer after first run, GPU "warmup"
		if (frame == 0)
			cudaTimerStart();
	}

	checkCudaErrors( hipDeviceSynchronize() );

	std::cout << cudaTimerStop() / (num_frames - 1) << "ms" << std::endl;

	if (argc == 2)
	{
		for (int slice = 0; slice < NUM_SLICES; slice++)
		{
			imshow(cv_gpu::GpuMat(N, N, CV_32FC1, out_buffer + N*N*slice));
		}
	}

	// TODO: reimplement cleanup code once satisfied with implementation

	return 0;
}
