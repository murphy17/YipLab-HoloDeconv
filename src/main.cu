#include "hip/hip_runtime.h"

/*
 *
 * Proof-of-concept for GPU holographic deconvolution.
 * Michael Murphy, May 2017
 * Yip Lab
 *
 */

// FFT batching
// async memory transfer
// half-precision
// texture memory

// microoptimisation: replace x * N with x << LOG2N

// should make some wrappers for the kernels, would make some decisions clearer

// This fails to run with 5.2 CC...

#include <opencv2/opencv.hpp>
#include <opencv2/highgui.hpp>
//#include <opencv2/gpu/gpu.hpp>
#include <opencv2/core/cuda.hpp>
#include <hip/hip_runtime.h>
#include <hipfft/hipfftXt.h>
#include <algorithm>
#include <hip/hip_fp16.h>

#include "common.h"

#define N 1024
#define LOG2N 10
#define DX (5.32f / 1024.f)
#define DY (6.66f / 1280.f)
#define LAMBDA0 0.000488f
#define SCALE 0.00097751711f // 1/(N-1)

typedef unsigned char byte;

// Convenience method for plotting
void imshow(cv::Mat in)
{
	cv::namedWindow("Display window", cv::WINDOW_NORMAL); // Create a window for display.
	cv::Mat out;
	hipDeviceSynchronize();
	in.convertTo(out, CV_32FC1);
	cv::normalize(out, out, 1.0, 0.0, cv::NORM_MINMAX, -1);
	cv::imshow("Display window", out); // Show our image inside it.
	cv::waitKey(0);
}
void imshow(cv::cuda::GpuMat in)
{
	cv::namedWindow("Display window", cv::WINDOW_NORMAL); // Create a window for display.
	cv::Mat out;
	hipDeviceSynchronize();
	in.download(out);
	if (out.channels() == 2)
	{
		cv::Mat channels[2];
		cv::split(out, channels);
		cv::magnitude(channels[0], channels[1  ], out);
	}
	out.convertTo(out, CV_32FC1);
	cv::normalize(out, out, 1.0, 0.0, cv::NORM_MINMAX, -1);
	cv::imshow("Display window", out); // Show our image inside it.
	cv::waitKey(0);
}

// Kernel to construct the point-spread function at distance z.
// exploits 4-fold symmetry (PSF is also radially symmetric, but that's harder...)
// note that answer is scaled between +/-1
__global__
void construct_psf(float z, hipfftComplex *g, float norm)
{
	const int i = blockIdx.x;
	const int j = threadIdx.x; // blockDim shall equal N

	const int ii = (N - 1) - i;
	const int jj = (N - 1) - j;

	// not sure whether the expansion of N/(N-1) was necessary
	float x = (i * SCALE + i - N/2) * DX;
	float y = (j * SCALE + j - N/2) * DY;

	// could omit negation here, symmetries of trig functions take care of it
	float r = (-2.f / LAMBDA0) * norm3df(x, y, z);

	// exp(ix) = cos(x) + isin(x)
	float re, im;
	sincospif(r, &im, &re);

	// numerical conditioning, important for half-precision FFT
	// also corrects the sign flip above
	r = __fdividef(r, norm); // norm = -2.f * z / LAMBDA0

	// re(iz) = -im(z), im(iz) = re(z)
	hipfftComplex g_ij;
	g_ij.x = __fdividef(-im, r); // im, r);
	g_ij.y = __fdividef(re, r);

	// CUDA takes care of coalescing the reversed access, this is fine
	g[i*N+j] = g_ij;
	g[i*N+jj] = g_ij;
	g[ii*N+j] = g_ij;
	g[ii*N+jj] = g_ij;
}

// exploit Fourier duality to shift without copying
// credit to http://www.orangeowlsolutions.com/archives/251
__global__
void frequency_shift(hipfftComplex *data)
{
    const int i = blockIdx.x;
    const int j = threadIdx.x;

	const float a = 1 - 2 * ((i+j) & 1); // this looks like a checkerboard?

	data[i*N+j].x *= a;
	data[i*N+j].y *= a;
}

__device__ __forceinline__
void _mul(void *dataOut, size_t offset, hipfftComplex a, void *callerInfo, void *sharedPtr)
{
	float bx = ((hipfftComplex *)callerInfo)[offset].x;
	float by = ((hipfftComplex *)callerInfo)[offset].y;

	float a_temp = a.y;
	float ay_by = __fmul_rn(a_temp, by);
	float ay_bx = __fmul_rn(a_temp, bx);
	a_temp = a.x;
	bx = __fmaf_rn(a_temp, bx, -ay_by);
	by = __fmaf_rn(a_temp, by, ay_bx);

	((hipfftComplex *)dataOut)[offset].x = bx;
	((hipfftComplex *)dataOut)[offset].y = by;
}
__device__
hipfftCallbackStoreC d_mul = _mul;

__global__
void byte_to_complex(byte *b, hipfftComplex *z)
{
	const int i = blockIdx.x;
	const int j = threadIdx.x; // blockDim shall equal N

	z[i*N+j].x = ((float)(b[i*N+j])) / 255.f;
	z[i*N+j].y = 0.f;
}

__global__
void byte_to_half2(byte *b, half2 *z)
{
	int i = blockIdx.x;
	int j = threadIdx.x; // blockDim shall equal N

	z[i*N+j] = __floats2half2_rn(((float)(b[i*N+j])) / 255.f, 0.f);
}

__global__
void complex_to_half2(hipfftComplex *z, half2 *h)
{
	int i = blockIdx.x;
	int j = threadIdx.x; // blockDim shall equal N

	h[i*N+j] = __float22half2_rn(z[i*N+j]);
}

__global__
void half2_to_complex(half2 *h, hipfftComplex *z)
{
	int i = blockIdx.x;
	int j = threadIdx.x; // blockDim shall equal N

	z[i*N+j] = __half22float2(h[i*N+j]);
}

__global__
void normalize_by(hipfftComplex *z, float n)
{
	int i = blockIdx.x;
	int j = threadIdx.x; // blockDim shall equal N

	z[i*N+j].x /= n;
	z[i*N+j].y /= n;
}

__global__
void normalize_by(half2 *h, float n)
{
	int i = blockIdx.x;
	int j = threadIdx.x; // blockDim shall equal N

	h[i*N+j] = __hmul2(h[i*N+j], __float2half2_rn(1.f / n));
}

__global__
void complex_modulus(hipfftComplex *z, float *r)
{
	const int i = blockIdx.x;
	const int j = threadIdx.x; // blockDim shall equal N

	r[i*N+j] = hypotf(z[i*N+j].x, z[i*N+j].y);
}

int main(int argc, char* argv[])
{
	checkCudaErrors( hipDeviceReset() );

	long long dims[] = {N, N};
	size_t work_sizes = 0;
	hipfftHandle plan, plan_mul;
	hipfftCreate(&plan);
	hipfftCreate(&plan_mul);
	checkCudaErrors( hipfftXtMakePlanMany(plan, 2, dims, \
			NULL, 1, 0, HIP_C_16F, \
			NULL, 1, 0, HIP_C_16F, \
			1, &work_sizes, HIP_C_16F) );

	hipfftComplex *d_img;
	checkCudaErrors( hipMalloc((void **)&d_img, N*N*sizeof(hipfftComplex)) );

	byte *d_img_u8;
	checkCudaErrors( hipMalloc((void **)&d_img_u8, N*N*sizeof(byte)) );

	half2 *d_img_f16;
	checkCudaErrors( hipMalloc((void **)&d_img_f16, N*N*sizeof(half2)) );

	cv::Mat A = cv::imread("test_square.bmp", CV_LOAD_IMAGE_GRAYSCALE);

	checkCudaErrors( hipMemcpy(d_img_u8, A.data, N*N*sizeof(byte), hipMemcpyHostToDevice) );

	// from symmetry alone, 1/N would make sense... (sqrt of N*N)

	// need to figure out normalization here
	// normalize image to [0,1] ... max pixel value is 1
	// min nonzero pixel value is 1/255
	// |Fourier coefficients| <= N*N

	byte_to_complex<<<N, N>>>(d_img_u8, d_img);
	complex_to_half2<<<N, N>>>(d_img, d_img_f16);

	// this works! division by N is dumb heuristic though
	// Parseval's theorem?
	normalize_by<<<N, N>>>(d_img_f16, N);

	checkCudaErrors( hipfftXtExec(plan, d_img_f16, d_img_f16, HIPFFT_FORWARD) );

	normalize_by<<<N, N>>>(d_img_f16, N);

	checkCudaErrors( hipfftXtExec(plan, d_img_f16, d_img_f16, HIPFFT_BACKWARD) );

	half2_to_complex<<<N, N>>>(d_img_f16, d_img);

	imshow(cv::cuda::GpuMat(N, N, CV_32FC2, d_img));

	return 0;
}
