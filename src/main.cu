#include "hip/hip_runtime.h"
/*
 *
 * Proof-of-concept for GPU holographic deconvolution.
 * Michael Murphy, May 2017
 * Yip Lab
 *
 */

// FFT batching
// async memory transfer
// half-precision
// texture memory

// This fails to run with 5.2 CC...

#include <opencv2/opencv.hpp>
#include <opencv2/highgui.hpp>
#include <hip/hip_runtime.h>
#include <hipfft/hipfftXt.h>

#include "common.h"

#define N 1024
#define DX (5.32 / 1024)
#define DY (6.66 / 1280)
#define LAMBDA0 0.000488

// Convenience method for plotting
void imshow(cv::Mat in)
{
	cv::namedWindow("Display window", cv::WINDOW_NORMAL); // Create a window for display.
	cv::Mat out;
	in.convertTo(out, CV_32FC1);
	cv::normalize(out, out, 1.0, 0.0, cv::NORM_MINMAX, -1);
	cv::imshow("Display window", out); // Show our image inside it.
	cv::waitKey(0);
}
void imshow(cv::cuda::GpuMat in)
{
	cv::namedWindow("Display window", cv::WINDOW_NORMAL); // Create a window for display.
	cv::Mat out;
	hipDeviceSynchronize();
	in.download(out);
	if (out.channels() == 2)
	{
		cv::Mat channels[2];
		cv::split(out, channels);
		cv::magnitude(channels[0], channels[1  ], out);
	}
	out.convertTo(out, CV_32FC1);
	cv::normalize(out, out, 1.0, 0.0, cv::NORM_MINMAX, -1);
	cv::imshow("Display window", out); // Show our image inside it.
	cv::waitKey(0);
}

// Kernel to construct the point-spread function at distance z.
// exploits 4-fold symmetry (PSF is also radially symmetric, but that's harder...)
// note that answer is scaled between +/-1
__global__
void construct_psf_4fold(float z, hipfftComplex *g)
{
	int i = blockIdx.x;
	int j = threadIdx.x; // blockDim shall equal N
	int ii = (N - 1) - i;
	int jj = (N - 1) - j;

	float x = (i * (float)N/(float)(N-1) - N/2) * DX;
	float y = (j * (float)N/(float)(N-1) - N/2) * DY;

	float r = (-2.f / LAMBDA0) * norm3df(x, y, z);

	// cos(x) = cos(-x), -sin(x) = sin(-x)
	// exp(ix) = cos(x) + isin(x)
	float re, im;
	sincospif(r, &im, &re);

	// !!! the normalization and sign-flip add 15ms to the runtime here -- related to register usage

	// numerical conditioning, important for half-precision FFT
	// also corrects the sign flip above
	r /= -2.f * z / LAMBDA0;

	// re(iz) = -im(z), im(iz) = re(z)
	hipfftComplex g_ij;
	g_ij.x = -im / r;
	g_ij.y = re / r;

	g[i*N+j] = g_ij;
	g[i*N+jj] = g_ij;
	g[ii*N+j] = g_ij;
	g[ii*N+jj] = g_ij;

	// this is slower!?!?!?!?!
	// write the half-row
//	g[i*N+j] = g_ij;
//	g[((N - 1) - i)*N+j] = g_ij;
//	// flip the half-row
//	g_r[(N/2 - 1) - j] = g_ij; // bank conflicts? can't avoid I think
//	__syncthreads(); // needed?
//	// write the flipped half-row
//	g[i*N+j+N/2] = g_r[j];
//	g[((N - 1) - i)*N+j+N/2] = g_r[j];
}

// exploit Fourier duality to shift without copying
// credit to http://www.orangeowlsolutions.com/archives/251
__global__
void frequency_shift(hipfftComplex *data)
{
    int i = blockIdx.x;
    int j = threadIdx.x;

	float a = 1 - 2 * ((i+j) & 1);

	data[i*N+j].x *= a;
	data[i*N+j].y *= a;
}

// it seems you can't have too many plans simultaneously?
// workaround: conditionals in the callback?

__device__
void _mul(void *dataOut, size_t offset, hipfftComplex element, void *callerInfo, void *sharedPtr)
{
	hipfftComplex a, b, c;

	a = element;
	b = ((hipfftComplex *)callerInfo)[offset];

	c.x = a.x * b.x - a.y * b.y;
	c.y = a.x * b.y + a.y * b.x;

	((hipfftComplex *)dataOut)[offset] = c;
}
__device__
hipfftCallbackStoreC d_mul = _mul;

__global__
void byte_to_complex(unsigned char *b, hipfftComplex *z)
{
	int i = blockIdx.x;
	int j = threadIdx.x; // blockDim shall equal N

	z[i*N+j].x = ((float)(b[i*N+j])) / 255.f;
	z[i*N+j].y = 0.f;
}

__global__
void complex_mod(hipfftComplex *z, float *r)
{
	int i = blockIdx.x;
	int j = threadIdx.x; // blockDim shall equal N

	r[i*N+j] = hypotf(z[i*N+j].x, z[i*N+j].y);
}

//__device__
//void _mod(void *dataOut, size_t offset, hipfftComplex element, void *callerInfo, void *sharedPtr)
//{
//	((float *)dataOut)[offset] = hypotf(element.x, element.y);
//}
//__device__
//hipfftCallbackStoreC d_mod = _mod;

int main(void)
{
	checkCudaErrors( hipDeviceReset() );

	int num_frames = 1;
	int num_slices = 100;
	float z_min = 30;
	float z_step = 1;

	hipStream_t math_stream, copy_stream;
	checkCudaErrors( hipStreamCreate(&math_stream) );
	checkCudaErrors( hipStreamCreate(&copy_stream) );

	hipfftHandle plan, plan_mul;
	checkCudaErrors( hipfftPlan2d(&plan, N, N, HIPFFT_C2C) );
//	checkCudaErrors( hipfftPlan2d(&plan_inv, N, N, HIPFFT_C2C) );
	checkCudaErrors( hipfftPlan2d(&plan_mul, N, N, HIPFFT_C2C) );
//	checkCudaErrors( hipfftPlan2d(&plan_mod, N, N, HIPFFT_C2C) );

//	checkCudaErrors( hipfftSetStream(plan_inv, math_stream) );
	checkCudaErrors( hipfftSetStream(plan_mul, math_stream) );
//	checkCudaErrors( hipfftSetStream(plan_mod, math_stream) );

	hipfftComplex *d_img, *d_psf, *d_psf_2;
	checkCudaErrors( hipMalloc((void **)&d_psf, N*N*sizeof(hipfftComplex)) );
	checkCudaErrors( hipMalloc((void **)&d_img, N*N*sizeof(hipfftComplex)) );
	checkCudaErrors( hipMalloc((void **)&d_psf_2, N*N*sizeof(hipfftComplex)) );

	hipfftCallbackStoreC h_mul;
	checkCudaErrors( hipMemcpyFromSymbol(&h_mul, HIP_SYMBOL(d_mul), sizeof(hipfftCallbackStoreC)) );
	checkCudaErrors( hipfftXtSetCallback(plan_mul, (void **)&h_mul, HIPFFT_CB_ST_COMPLEX, (void **)&d_img) );

//	hipfftCallbackStoreC h_mod;
//	checkCudaErrors( hipMemcpyFromSymbol(&h_mod, HIP_SYMBOL(d_mod), sizeof(hipfftCallbackStoreC)) );
//	checkCudaErrors( hipfftXtSetCallback(plan_mod, (void **)&h_mod, HIPFFT_CB_ST_COMPLEX, 0) );

	unsigned char *d_img_u8;
	checkCudaErrors( hipMalloc((void **)&d_img_u8, N*N*sizeof(unsigned char)) );

	float *d_slices;
	checkCudaErrors( hipMalloc((void **)&d_slices, num_slices*N*N*sizeof(float)) );

	float *h_slices;
	checkCudaErrors( hipHostMalloc((void **)&h_slices, num_slices*N*N*sizeof(float)) );

	for (int frame = 0; frame < num_frames; frame++)
	{
		// this would be a copy from a frame buffer on the Jetson
		cv::Mat A = cv::imread("test_square.bmp", CV_LOAD_IMAGE_GRAYSCALE);

		cudaTimerStart();

		checkCudaErrors( hipMemcpy(d_img_u8, A.data, N*N*sizeof(unsigned char), hipMemcpyHostToDevice) );

		byte_to_complex<<<N, N>>>(d_img_u8, d_img);

		checkCudaErrors( hipfftExecC2C(plan, d_img, d_img, HIPFFT_FORWARD) );
		hipStreamSynchronize(math_stream); // reusing a plan

		// this is subtle - shifting in conjugate domain means we don't need to FFT shift later
		// obvious - simple - callback candidate, but would be waste of time
		frequency_shift<<<N, N>>>(d_img);

		// definitely BATCHING is next big speedup
		for (int slice = 0; slice < num_slices; slice++)
		{
			float z = z_min + z_step * slice;

			// faster to do this than the callback, curiously... need to further investigate shared memory
			// !!! see if register count can be dropped; and
			// !!! play with thread/block count: more blocks, fewer threads?
			construct_psf_4fold<<<N/2, N/2, 0, math_stream>>>(z, d_psf);

			// investigate in vs out of place
			// FFT and multiply
			checkCudaErrors( hipfftExecC2C(plan_mul, d_psf, d_psf, HIPFFT_FORWARD) ); // big speedup with callback! 1.4ms -> 0.8ms

			checkCudaErrors( hipfftExecC2C(plan, d_psf, d_psf, HIPFFT_BACKWARD) ); // doing the mod in here shaves off ~15ms
			// for FFT shift would need to invert phase now, but it doesn't matter since we're taking modulus

			complex_mod<<<N, N, 0, math_stream>>>(d_psf, d_slices + N*N*slice);

			// I am utterly puzzled as to why this works but d_img_real didn't !!! BAD !!!
			// (suspect there's some sort of bounds checking that happens with the output)
			checkCudaErrors( hipMemcpyAsync(h_slices + N*N*slice, d_slices + N*N*slice, N*N*sizeof(float), hipMemcpyDeviceToHost, copy_stream) );
			// on a Tegra box the aforementioned copy wouldn't happen, the cufft would write direct to a cube

//			imshow(cv::cuda::GpuMat(N, N, CV_32FC1, d_mod));
		}

		// zero-copy memory in Tegra
//		checkCudaErrors( hipMemcpy(h_slices, d_slices, num_slices*N*N*sizeof(float), hipMemcpyDeviceToHost) );

		std::cout << cudaTimerStop() << "ms" << std::endl;
		// 145ms with async copies + stupid d_psf thing, AF was something like 125ms... closing in...
		// looking at utilisation, might be able to halve (!!!) that with batching
	}

	// hipFree...

	checkCudaErrors( hipfftDestroy(plan) );
	checkCudaErrors( hipfftDestroy(plan_mul) );
//	checkCudaErrors( hipfftDestroy(plan_mod) );

	checkCudaErrors( hipStreamDestroy(math_stream) );
	checkCudaErrors( hipStreamDestroy(copy_stream) );

	return 0;
}

/*

    // load in test image
    cv::Mat A = cv::imread("test_square.bmp", CV_LOAD_IMAGE_GRAYSCALE);
    A.convertTo(A, CV_32FC1);
//    imshow(A);

    // relevant! http://arrayfire.com/zero-copy-on-tegra-k1/
    // hipSetDeviceFlags(hipDeviceMapHost);

	complex *g, *h;
//	float *f;
	hipMalloc((void **)&g, N*N*sizeof(complex));
	hipMalloc((void **)&h, N*N*sizeof(complex));
//	hipMalloc((void **)&f, N*N*sizeof(float));

	// these two take a long time...
	float *R_d, *R_h;
	hipHostMalloc((void **)&R_h, num_slices*N*N*sizeof(float));
	hipMalloc((void **)&R_d, num_slices*N*N*sizeof(float));

	float z_h[num_slices];
	float *z_d;
	hipMalloc((void **)&zs, num_slices*sizeof(float));
	for (int k = 0; k < num_slices; k++) z_h[k] = z_min + dz*k;
	hipMemcpy(z_d, z_h, num_slices*sizeof(float), hipMemcpyHostToDevice);
	// now pass this to callback

	hipfftHandle plan, plan_mul, plan_psf;
	hipfftPlan2d(&plan, N, N, HIPFFT_C2C); // cufftplanmany for batch...
	hipfftPlan2d(&plan_psf, N, N, HIPFFT_C2C);
	hipfftPlan2d(&plan_mul, N, N, HIPFFT_C2C);
	hipfftXtSetCallback(plan_psf, (void **)&_psf_ptr, HIPFFT_CB_ST_COMPLEX, (void **)&h);
	hipfftXtSetCallback(plan_mul, (void **)&_mul_ptr, HIPFFT_CB_ST_COMPLEX, (void **)&h);

//	construct_psf_4fold<<<N/2, N/2>>>(50, g); // 0.40s for 100
//	mod<<<N, N>>>(g, f); // 0.20s for 100
//	cv::Mat B(N, N, CV_32FC1);
//	hipMemcpy(B.data, f, N*N*sizeof(float), hipMemcpyDeviceToHost);
//	imshow(B);

	// some behaviour I can't explain here.
	// results look great the first time
	// second time... output is corrupted (!?!)
	for (int n_frame = 0; n_frame < num_frames; n_frame++)
	{
		// transfer image to device, using padding to add imaginary channel
		 hipMemcpy2D(h, sizeof(complex), A.data, sizeof(float), sizeof(float), N*N, hipMemcpyHostToDevice);

		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start);

		hipfftExecC2C(plan, (hipfftComplex *)h, (hipfftComplex *)h, HIPFFT_FORWARD);
		// this is subtle - shifting in conjugate domain means we don't need to FFT shift later
		frequency_shift<<<N, N>>>(h);

		float ms = 0;

		for (int k = 0; k < num_slices; k++)
		{
			float z = z_min + dz*k;

			// performance seems better with more blocks, fewer threads
			// this actually has 8-fold symmetry. rotate a symmetric matrix 4 times about the origin
			construct_psf_4fold<<<N/2, N/2>>>(z, g); // 0.40s for 100

			hipfftExecC2C(plan, (hipfftComplex *)g, (hipfftComplex *)g, HIPFFT_FORWARD); // 0.28s for 100

//			multiply_inplace<<<N, N>>>(g, h); // 1.38s for 100!?! why so slow???

//			hipEvent_t start, stop;
//			hipEventCreate(&start);
//			hipEventCreate(&stop);
//			hipEventRecord(start);

			hipfftExecC2C(plan_mul, (hipfftComplex *)g, (hipfftComplex *)g, HIPFFT_BACKWARD); // 0.28s for 100
			// ordinarily would need to invert phase now, to compensate for earlier (and complete the FFT shift)...

//			hipDeviceSynchronize();
//			hipEventRecord(stop);
//			hipEventSynchronize(stop);
//			float ms_ = 0;
//			hipEventElapsedTime(&ms_, start, stop);
//			ms += ms_;

			// ... but since we're just taking the modulus, we don't care about phase
			complex_mod<<<N, N>>>(g, R_d + N*N*k); // 0.20s for 100
		}

		// hipDestroyTextureObject(tex);

		// do some reduction on the images
		// ...

		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&ms, start, stop);

		std::cout << ms << "ms" << std::endl;

		hipMemcpy(R_h, R_d, num_slices*N*N*sizeof(float), hipMemcpyDeviceToHost);
	}

	// free pointers
	hipFree(g);
	hipFree(h);
	hipFree(R_d);

	for (int k = 0; k < num_slices; k++)
	{
		cv::Mat B(N, N, CV_32FC1, R_h + k*N*N);
		imshow(B);
	}

	hipFree(R_h);

	return 0;
}

/*

typedef struct {
// int N;
float lambda0;
float del_x;
float del_y;
float d_max;
float d_min;
float d_step;
int n_batch;
int n_frames;
} params_t;

// Populates frequency grid and vector of spacings.
//void setup_vars(params_t params, af::array *grid, af::array *spacings)
//{
// int n_step = (params.d_max - params.d_min) / params.d_step;
//    // build (symmetric?) path-length grid
// af::array df = (af::iota(af::dim4(N, 1)) * (float)(N)/(float)(N-1)-N/2);
//    *grid = af::tile(af::pow(df * params.del_x, 2), 1, N) + af::tile(af::pow(df * params.del_y, 2).T(), N, 1);
//    *spacings = af::pow(af::range(af::dim4(n_step)) * params.d_step + params.d_min, 2);
//}

// Expands input image into slices, performs a reduction, and writes the result out.
void process_image(params_t params, af::array &img, float *out_ptr, af::array &x_cube) //, af::array &grid, af::array &spacings)
{
af::array x(N, N, c32);

// int n_step = spacings.dims(0);
//    af::cfloat k0 = {0., (float)(-2. * af::Pi / params.lambda0)};
//    af::cfloat k1 = {0., (float)(1. / params.lambda0)};
af::cfloat unit = {0, 1};

// FFT the input image
af::array h_f = af::fft2(img);
// phase shift it
h_f = h_f * unit;

int n_step = (params.d_max - params.d_min) / params.d_step;

// process in batches to fit in memory
// ... but this seems to entirely occupy the Tegra...
for (int j = 0; j < n_step; j ++)
{
float z = params.d_min + j * params.d_step;
af::sync();
complex *d_x = (complex *)x.device<af::cfloat>();
construct_psf_4fold<<<N/2, N/2>>>(z, d_x);
hipDeviceSynchronize();
x.unlock();
// x = af::sqrt(grid + params.d_min + i * params.d_step); // ~0.15sec
// x = k1 * af::exp(k0 * x) / x; // g
af::fft2InPlace(x); // g_f // 0.3sec

// note here: g is an even function
// so F(g) is real valued and even
// (i.e. can I just take FFT of half of g?)

x = x * h_f; // gh_f
af::ifft2InPlace(x); // h // 0.4sec
// x = af::shift(x, x.dims(0)/2, x.dims(1)/2); // FFT shift

x_cube(af::span, af::span, j) = af::abs(x); // / af::max<float>(x) * 255.; // compression probably unnecessary?
}

// simulate doing some reduction operation that returns a single image per cube
// i.e. find optimal focus -> construct 3D volume
af::array x_sum = af::sum(x_cube, 2);

// push to host
x_sum.host(out_ptr);
}

int main(void)
{
// setup experimental parameters
    params_t params;
//    N = 1024; // resolution in pixels
    params.lambda0 = 0.000488; // wavelength
    params.del_x = 5.32 / 1024; // horizontal frequency spacing
    params.del_y = 6.66 / 1280; // vertical frequency spacing
    params.d_max = 130; // max distance from CCD to object, mm
    params.d_min = 30; // min distance from CCD to object, mm
    params.d_step = 1; // step size in mm
    int N_batch = 1; // number of frames per batch; best performance with 1!?!?
    int N_frames = 3; // currently, just repeats the analysis

    // load in test image
    cv::Mat mat = cv::imread("test_square.bmp", CV_LOAD_IMAGE_GRAYSCALE);
    mat.convertTo(mat, CV_32FC1);

    // simulate a DMA buffer on the GPU, i.e. feeding in from video camera
    // will copy in the images as part of the main loop, simulate 'streaming'
    float *img_ptr;
    hipMalloc((void **)&img_ptr, N * N * sizeof(float));

    // allocate the matrix using our predefined staging area
    af::array img(N, N, img_ptr, afDevice);
    af::eval(img);

    // pin buffer on the host
    float *h_ptr = af::pinned<float>(N * N * N_frames);

//    af::array grid;
//    af::array spacings;
//    setup_vars(params, &grid, &spacings);

    // allocate this just once and reuse, it's huge
//    int n_step = spacings.dims(0);
    int n_step = (params.d_max - params.d_min) / params.d_step;
    af::array x_cube(N, N, n_step, f32);

    for (int k = 0; k < N_frames; k++)
    {
// 'copy the image' - these would be successive frames in reality, and would probably live on GPU
// i.e. this copy would not happen
// mat = mat + 0.; // no possibility of caching
hipMemcpy(img_ptr, mat.data, N * N * sizeof(float), hipMemcpyHostToDevice);

// expand the image into slices, do a reduction, save result to h_ptr
af::timer::start();
process_image(params, img, h_ptr + N * N * k, x_cube); //, grid, spacings);
std::cout << af::timer::stop() << std::endl;
    }

    cv::namedWindow("Display window", cv::WINDOW_NORMAL); // Create a window for display.
    for (int i = 0; i < n_step; i++)
    {
        cv::Mat mat(cv::Size(1024, 1024), CV_32FC1, h_ptr + i * N * N);
        cv::normalize(mat, mat, 1.0, 0.0, cv::NORM_MINMAX, -1);
        cv::imshow("Display window", mat); // Show our image inside it.
        cv::waitKey(0);
    }

    return 0;
}

*/
